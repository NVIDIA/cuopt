/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../linear_programming/utilities/pdlp_test_utilities.cuh"
#include "mip_utils.cuh"

#include <cuopt/linear_programming/mip/solver_solution.hpp>
#include <linear_programming/pdlp.cuh>
#include <linear_programming/utilities/problem_checking.cuh>
#include <mip/presolve/trivial_presolve.cuh>
#include <mip/relaxed_lp/relaxed_lp.cuh>
#include <mps_parser/parser.hpp>
#include <utilities/common_utils.hpp>
#include <utilities/error.hpp>

#include <raft/sparse/detail/cusparse_wrappers.h>
#include <raft/core/handle.hpp>
#include <raft/util/cudart_utils.hpp>

#include <gtest/gtest.h>

#include <thrust/count.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>

#include <cstdint>
#include <limits>
#include <sstream>
#include <string>
#include <vector>

namespace cuopt::linear_programming::test {

static std::tuple<mip_termination_status_t, double, double> test_mps_file(std::string test_instance,
                                                                         bool heuristics_only = true,
                                                                         double time_limit    = 10)
{
  const raft::handle_t handle_{};

  auto path = make_path_absolute(test_instance);
  cuopt::mps_parser::mps_data_model_t<int, double> problem =
    cuopt::mps_parser::parse_mps<int, double>(path, false);
  handle_.sync_stream();
  mip_solver_settings_t<int, double> settings;
  settings.time_limit                  = time_limit;
  settings.heuristics_only             = heuristics_only;
  mip_solution_t<int, double> solution = solve_mip(&handle_, problem, settings);
  return std::make_tuple(solution.get_termination_status(),
                        solution.get_objective_value(),
                        solution.get_solution_bound());
}

TEST(termination_status, trivial_presolve_optimality_test)
{
  auto [termination_status, obj_val, lb] = test_mps_file("mip/trivial-presolve-optimality.mps");
  EXPECT_EQ(termination_status, mip_termination_status_t::Optimal);
  EXPECT_EQ(obj_val, -1);
}

TEST(termination_status, presolve_optimality_test)
{
  auto [termination_status, obj_val, lb] = test_mps_file("mip/sudoku.mps");
  EXPECT_EQ(termination_status, mip_termination_status_t::Optimal);
  EXPECT_EQ(obj_val, 0);
}

TEST(termination_status, presolve_infeasible_test)
{
  auto [termination_status, obj_val, lb] = test_mps_file("mip/presolve-infeasible.mps");
  EXPECT_EQ(termination_status, mip_termination_status_t::Infeasible);
}

TEST(termination_status, feasible_found_test)
{
  auto [termination_status, obj_val, lb] = test_mps_file("mip/gen-ip054.mps");
  EXPECT_EQ(termination_status, mip_termination_status_t::FeasibleFound);
}

TEST(termination_status, timeout_test)
{
  auto [termination_status, obj_val, lb] = test_mps_file("mip/stein9inf.mps");
  EXPECT_EQ(termination_status, mip_termination_status_t::TimeLimit);
}

TEST(termination_status, optimality_test)
{
  auto [termination_status, obj_val, lb] = test_mps_file("mip/bb_optimality.mps", false);
  EXPECT_EQ(termination_status, mip_termination_status_t::Optimal);
  EXPECT_EQ(obj_val, 2);
}

// Ensure the lower bound on maximization problems when BB times out has the right sign
TEST(termination_status, lower_bound_bb_timeout)
{
  auto [termination_status, obj_val, lb] = test_mps_file("mip/cod105_max.mps", false, 0.5);
  EXPECT_EQ(termination_status, mip_termination_status_t::FeasibleFound);
  EXPECT_EQ(obj_val, 12);
  EXPECT_GE(lb, obj_val);
}

TEST(termination_status, bb_infeasible_test)
{
  // First, check that presolve doesn't reduce the problem to infeasibility
  {
    auto [termination_status, obj_val, lb] = test_mps_file("mip/stein9inf.mps", true, 1);
    EXPECT_EQ(termination_status, mip_termination_status_t::TimeLimit);
  }
  // Ensure that B&B proves the MIP infeasible
  {
    auto [termination_status, obj_val, lb] = test_mps_file("mip/stein9inf.mps", false, 30);
    EXPECT_EQ(termination_status, mip_termination_status_t::Infeasible);
  }
}

}  // namespace cuopt::linear_programming::test
