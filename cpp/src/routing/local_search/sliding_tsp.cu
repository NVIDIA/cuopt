#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/cuda_helpers.cuh>
#include "../solution/solution.cuh"
#include "../utilities/cuopt_utils.cuh"
#include "local_search.cuh"

namespace cuopt {
namespace routing {
namespace detail {

auto constexpr const max_window_size = 20;

template <typename i_t, typename f_t, request_t REQUEST>
DI thrust::pair<double, double> eval_move(
  typename solution_t<i_t, f_t, REQUEST>::view_t& sol,
  typename move_candidates_t<i_t, f_t>::view_t& move_candidates,
  const typename route_t<i_t, f_t, REQUEST>::view_t& s_route,
  raft::device_span<double> sh_reverse_dist,
  i_t intra_idx,
  i_t insertion_pos,
  i_t window_size,
  bool reverse)
{
  auto original_window_dist =
    s_route.dimensions.distance_dim.distance_forward[intra_idx + window_size - 1] -
    s_route.dimensions.distance_dim.distance_forward[intra_idx];
  auto new_window_dist =
    reverse ? sh_reverse_dist[intra_idx] - sh_reverse_dist[intra_idx + window_size - 1]
            : original_window_dist;

  auto original_previous_intra_frag_next =
    s_route.dimensions.distance_dim.distance_forward[intra_idx + window_size] -
    s_route.dimensions.distance_dim.distance_forward[intra_idx - 1];

  auto frag_begin = reverse ? intra_idx + window_size - 1 : intra_idx;
  auto frag_end   = reverse ? intra_idx : intra_idx + window_size - 1;
  auto insertion_pos_frag_begin =
    get_arc_of_dimension<i_t, f_t, dim_t::DIST>(s_route.get_node(insertion_pos).node_info(),
                                                s_route.get_node(frag_begin).node_info(),
                                                s_route.vehicle_info());

  // in-place
  if (insertion_pos == intra_idx - 1) {
    auto frag_end_frag_next = get_arc_of_dimension<i_t, f_t, dim_t::DIST>(
      s_route.get_node(frag_end).node_info(),
      s_route.get_node(intra_idx + window_size).node_info(),
      s_route.vehicle_info());
    auto delta = insertion_pos_frag_begin + new_window_dist + frag_end_frag_next -
                 original_previous_intra_frag_next;
    return {delta, delta};
  }

  auto frag_end_insertion_pos_next =
    get_arc_of_dimension<i_t, f_t, dim_t::DIST>(s_route.get_node(frag_end).node_info(),
                                                s_route.get_node(insertion_pos + 1).node_info(),
                                                s_route.vehicle_info());

  auto previous_intra_frag_next = get_arc_of_dimension<i_t, f_t, dim_t::DIST>(
    s_route.get_node(intra_idx - 1).node_info(),
    s_route.get_node(intra_idx + window_size).node_info(),
    s_route.vehicle_info());
  auto insertion_pos_insertion_pos_next =
    get_arc_of_dimension<i_t, f_t, dim_t::DIST>(s_route.get_node(insertion_pos).node_info(),
                                                s_route.get_node(insertion_pos + 1).node_info(),
                                                s_route.vehicle_info());
  auto delta = previous_intra_frag_next + insertion_pos_frag_begin + new_window_dist +
               frag_end_insertion_pos_next - insertion_pos_insertion_pos_next -
               original_previous_intra_frag_next;
  return {delta, delta};
}

template <typename i_t, typename f_t, request_t REQUEST>
__global__ void set_moved_regions_kernel(typename solution_t<i_t, f_t, REQUEST>::view_t sol,
                                         raft::device_span<i_t> moved_regions)
{
  auto route_id   = blockIdx.x;
  auto route      = sol.routes[route_id];
  auto max_active = sol.get_max_active_nodes_for_all_routes();
  for (i_t i = threadIdx.x; i < route.get_num_nodes(); i += blockDim.x) {
    moved_regions[route_id * max_active + i] = route.get_node(i).node_info().node();
  }
}

template <typename i_t, typename f_t, request_t REQUEST>
__global__ void find_sliding_moves_tsp(
  typename solution_t<i_t, f_t, REQUEST>::view_t sol,
  typename move_candidates_t<i_t, f_t>::view_t move_candidates,
  raft::device_span<sliding_tsp_cand_t<i_t>> sampled_nodes_data,
  raft::device_span<i_t> locks)
{
  extern __shared__ double shmem[];

  const i_t node_idx   = blockIdx.x;
  const auto node_info = move_candidates.nodes_to_search.sampled_nodes_to_search[node_idx];
  cuopt_assert(
    node_info.node() < sol.get_num_orders() + sol.n_routes * after_depot_insertion_multiplier,
    "Invalid node id");
  // special node that represent after depot insertion is ignored
  if (node_info.node() >= sol.get_num_orders()) { return; }

  // Retrive associated node info

  const auto [route_id, intra_idx] =
    sol.route_node_map.get_route_id_and_intra_idx(node_info.node());

  if (route_id == -1)  // Handle unrouted node case for GES
    return;

  cuopt_assert(route_id >= 0, "Invalid route id");
  cuopt_assert(route_id < sol.n_routes, "Invalid route id");

  auto route = sol.routes[route_id];

  auto s_route = route_t<i_t, f_t, REQUEST>::view_t::create_shared_route(
    (i_t*)shmem, route, route.get_num_nodes());
  __syncthreads();
  s_route.copy_from(route);
  __syncthreads();

  // reverse and non reverse frag
  auto sh_reverse_dist = raft::device_span<double>(
    reinterpret_cast<double*>(raft::alignTo(s_route.shared_end_address(), sizeof(double))),
    s_route.get_num_nodes() + 1);
  __syncthreads();
  if (threadIdx.x == 0) {
    sh_reverse_dist[s_route.get_num_nodes()] = 0.;
    for (int i = s_route.get_num_nodes() - 1; i >= 0; i--) {
      double dist = get_arc_of_dimension<i_t, f_t, dim_t::DIST>(s_route.get_node(i + 1).node_info(),
                                                                s_route.get_node(i).node_info(),
                                                                s_route.vehicle_info());
      sh_reverse_dist[i] = dist + sh_reverse_dist[i + 1];
    }
  }
  __syncthreads();

  const double excess_limit =
    s_route.get_weighted_excess(move_candidates.weights) * ls_excess_multiplier_route;

  sliding_tsp_cand_t<i_t> sliding_tsp_cand = is_sliding_tsp_uinitialized_t<i_t>::init_data;
  double cost_delta, selection_delta;

  auto n_reverse_types       = 2;
  auto n_insertion_pos       = s_route.get_num_nodes();
  auto route_max_window_size = max_window_size;  // s_route.get_num_nodes();
  auto total_permut          = route_max_window_size * n_reverse_types *
                      n_insertion_pos;  // forward, backward at every node pos
  for (i_t tid = threadIdx.x; tid < total_permut; tid += blockDim.x) {
    auto insertion_pos = tid % n_insertion_pos;
    auto frag_size     = tid / n_insertion_pos;
    auto window_size   = (frag_size % route_max_window_size) + 1;
    auto reverse       = frag_size / route_max_window_size;

    cuopt_assert(insertion_pos < s_route.get_num_nodes(), "Wrong insertion pos");
    // cuopt_assert(window_size <= route_max_window_size, "Wrong window_size");
    cuopt_assert(reverse == 0 || reverse == 1, "Wrong reverse val");

    if (intra_idx + window_size > s_route.get_num_nodes()) { continue; }

    // overlap
    if (!reverse && insertion_pos >= intra_idx - 1 && insertion_pos < intra_idx + window_size) {
      continue;
    }

    if (reverse && window_size == 1) { continue; }

    // authorize reverse in place (no -1)
    if (reverse && insertion_pos >= intra_idx && insertion_pos < intra_idx + window_size) {
      continue;
    }

    thrust::tie(cost_delta, selection_delta) = eval_move<i_t, f_t, REQUEST>(sol,
                                                                            move_candidates,
                                                                            s_route,
                                                                            sh_reverse_dist,
                                                                            intra_idx,
                                                                            insertion_pos,
                                                                            window_size,
                                                                            reverse);

    if (cost_delta > -EPSILON) { continue; }

    if (selection_delta < sliding_tsp_cand.selection_delta) {
      sliding_tsp_cand.insertion_pos   = insertion_pos;
      sliding_tsp_cand.window_size     = window_size;
      sliding_tsp_cand.window_start    = intra_idx;
      sliding_tsp_cand.reverse         = reverse;
      sliding_tsp_cand.selection_delta = selection_delta;
    }
  }

  // reduction
  __shared__ int reduction_index;
  __shared__ double shbuf[warp_size * 2];

  int idx = threadIdx.x;
  // block_reduce_ranked changes sliding_tsp_cand
  double saved_cost = sliding_tsp_cand.selection_delta;
  block_reduce_ranked(saved_cost, idx, shbuf, &reduction_index);

  if (threadIdx.x == reduction_index) { sampled_nodes_data[node_info.node()] = sliding_tsp_cand; }
}

template <typename i_t, typename f_t, request_t REQUEST>
DI void mark_impacted_nodes(const typename route_t<i_t, f_t, REQUEST>::view_t& route,
                            typename move_candidates_t<i_t, f_t>::view_t& move_candidates,
                            const sliding_tsp_cand_t<i_t>& best_candidate,
                            raft::device_span<i_t> moved_regions,
                            i_t n_orders,
                            i_t max_active)
{
  auto route_id = blockIdx.x;
  // mark the window itself and also the surrounding positions
  if (best_candidate.window_start - 1 == 0 || best_candidate.insertion_pos == 0) {
    if (threadIdx.x == 0) {
      move_candidates.nodes_to_search.active_nodes_impacted[route.get_id() + n_orders] = 1;
    }
  }
  // add two more nodes
  i_t start = max(best_candidate.window_start - 1, 1);
  // add two more nodes
  i_t end =
    min(best_candidate.window_start + best_candidate.window_size + 1, route.get_num_nodes());
  for (i_t i = threadIdx.x + start; i < end; i += blockDim.x) {
    cuopt_assert(moved_regions[route_id * max_active + i] != -1, "Node was already moved");
    move_candidates.nodes_to_search.active_nodes_impacted[route.node_id(i)] = 1;
    moved_regions[route_id * max_active + i]                                = -1;
  }

  start = max(best_candidate.insertion_pos, 1);
  end   = min(best_candidate.insertion_pos + 2, route.get_num_nodes());
  // mark the surroundings of the new position
  for (i_t i = threadIdx.x + start; i < end; i += blockDim.x) {
    move_candidates.nodes_to_search.active_nodes_impacted[route.node_id(i)] = 1;
    moved_regions[route_id * max_active + i]                                = -1;
  }
}

template <typename i_t, typename f_t, request_t REQUEST>
__global__ void execute_sliding_moves_tsp(
  typename solution_t<i_t, f_t, REQUEST>::view_t sol,
  typename move_candidates_t<i_t, f_t>::view_t move_candidates,
  raft::device_span<sliding_tsp_cand_t<i_t>> sampled_nodes_data,
  raft::device_span<i_t> moved_regions)
{
  extern __shared__ double shmem[];
  auto route_id = blockIdx.x;

  auto cand = sampled_nodes_data[0];
  if (cand.selection_delta == std::numeric_limits<double>::max()) { return; }

  auto max_active = sol.get_max_active_nodes_for_all_routes();
  auto route      = sol.routes[route_id];
  auto s_route    = route_t<i_t, f_t, REQUEST>::view_t::create_shared_route(
    (i_t*)shmem, route, route.get_num_nodes());
  __syncthreads();
  s_route.copy_from(route);
  __syncthreads();

  typename dimensions_route_t<i_t, f_t, REQUEST>::view_t fragment;
  i_t* dummy;
  // max_fragment_size-1, because the create shared route adds one more already
  thrust::tie(fragment, dummy) = dimensions_route_t<i_t, f_t, REQUEST>::view_t::create_shared_route(
    reinterpret_cast<i_t*>(raft::alignTo(s_route.shared_end_address(), sizeof(double))),
    sol.problem.dimensions_info,
    max_active - 1);
  __syncthreads();

  __shared__ i_t sh_overlaps;

  for (i_t x = 0; x < sampled_nodes_data.size(); ++x) {
    __syncthreads();

    if (threadIdx.x == 0) { sh_overlaps = 0; }
    __syncthreads();

    auto cand = sampled_nodes_data[x];

    if (cand.selection_delta == std::numeric_limits<double>::max()) { break; }

    // add two more nodes
    i_t start = max(cand.window_start - 1, 1);
    // add two more nodes
    i_t end = min(cand.window_start + cand.window_size + 1, s_route.get_num_nodes());
    for (i_t i = threadIdx.x + start; i < end; i += blockDim.x) {
      if (moved_regions[route_id * max_active + i] == -1) { sh_overlaps = 1; }
    }
    start = max(cand.insertion_pos - 1, 1);
    end   = min(cand.insertion_pos + 2, s_route.get_num_nodes());
    // mark the surroundings of the new position
    for (i_t i = threadIdx.x + start; i < end; i += blockDim.x) {
      if (moved_regions[route_id * max_active + i] == -1) { sh_overlaps = 1; }
    }
    __syncthreads();

    if (sh_overlaps) { continue; }

    cuopt_func_call(
      if (threadIdx.x == 0) { atomicAdd(move_candidates.debug_delta, cand.selection_delta); });

    auto original_node_id        = moved_regions[route_id * max_active + cand.window_start];
    auto original_node_insertion = moved_regions[route_id * max_active + cand.insertion_pos];
    cuopt_assert(original_node_id >= 0, "Moved region node id should be positive");
    cuopt_assert(original_node_insertion >= 0, "Moved region node id should be positive");

    mark_impacted_nodes<i_t, f_t, REQUEST>(
      route, move_candidates, cand, moved_regions, sol.get_num_orders(), max_active);
    __syncthreads();

    const auto [_, new_window_start] =
      sol.route_node_map.get_route_id_and_intra_idx(original_node_id);

    const auto [dummy, new_insertion_pos] =
      sol.route_node_map.get_route_id_and_intra_idx(original_node_insertion);
    cand.window_start  = new_window_start;
    cand.insertion_pos = original_node_insertion != 0 ? new_insertion_pos : 0;
    __syncthreads();

    fragment.parallel_copy_nodes_from(
      0, s_route, cand.window_start, cand.window_size, cand.reverse);
    __syncthreads();

    if (threadIdx.x == 0) {
      const int old_start_idx = cand.window_start;
      auto const window_size  = cand.window_size;
      cuopt_assert(old_start_idx > 0 && old_start_idx + window_size - 1 < s_route.get_num_nodes(),
                   "Invalid old start index");
      // Store
      const int new_start_idx = cand.insertion_pos;

      // In place insertion
      if (old_start_idx == new_start_idx) {
        for (int i = 0; i < window_size; ++i)
          s_route.set_node(i + new_start_idx, fragment.get_node(i));
      } else {
        // Left shift
        for (int i = old_start_idx; (i + window_size) < s_route.get_num_nodes(); ++i) {
          s_route.set_node(i, s_route.get_node(i + window_size));
        }
        // Right shift nodes to leave room for the window insertion

        cuopt_assert(new_start_idx >= 0 && new_start_idx < s_route.get_num_nodes(),
                     "Invalid new start index");
        for (int i = s_route.get_num_nodes() - 1;
             i > new_start_idx + ((new_start_idx < old_start_idx) ? window_size : 0);
             --i) {
          cuopt_assert(i - window_size >= 0, "Negative index");
          s_route.set_node(i, s_route.get_node(i - window_size));
        }
        // Insert the nodes

        // + 1 because we insert after the node
        // Handle case where insertion is after initial position of window
        for (int i = 0; i < window_size; ++i) {
          s_route.set_node(
            i + new_start_idx + 1 - ((new_start_idx > old_start_idx) ? window_size : 0),
            fragment.get_node(i));
        }
      }
    }
    __syncthreads();

    // Update intra_route_idx_per_node
    for (int i = threadIdx.x + 1; i < s_route.get_num_nodes(); i += blockDim.x) {
      const auto& node_info = s_route.requests().node_info[i];
      sol.route_node_map.set_intra_route_idx(node_info, i);
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    // Update info
    route_t<i_t, f_t, REQUEST>::view_t::compute_forward(s_route);
    route_t<i_t, f_t, REQUEST>::view_t::compute_backward(s_route);
    s_route.compute_cost();
    sol.routes_to_copy[route_id]   = 1;
    sol.routes_to_search[route_id] = 1;
  }
  __syncthreads();

  route.copy_from(s_route);
}

template <typename i_t, typename f_t, request_t REQUEST>
bool local_search_t<i_t, f_t, REQUEST>::perform_sliding_tsp(
  solution_t<i_t, f_t, REQUEST>& sol, move_candidates_t<i_t, f_t>& move_candidates)
{
  raft::common::nvtx::range fun_scope("perform_sliding_tsp");
  sol.global_runtime_checks(false, false, "sliding_tsp_start");
  i_t n_moves_found = 0;
  if (!move_candidates.include_objective) { return false; }
  [[maybe_unused]] double cost_before = 0., cost_after = 0.;

  auto constexpr const n_threads = 64;

  // if a single route size is greater than select a random subset of 500 nodes to search
  constexpr i_t tsp_route_size_threshold = 512;
  if (sol.get_num_depot_excluded_orders() > tsp_route_size_threshold) {
    fill_tsp_random_considered_nodes(sol, move_candidates, rng);
  }

  auto n_blocks = move_candidates.nodes_to_search.n_sampled_nodes;
  async_fill(
    sampled_tsp_data_, is_sliding_tsp_uinitialized_t<i_t>::init_data, sol.sol_handle->get_stream());

  auto is_tsp = sol.problem_ptr->is_tsp;

  auto shared_route_size = sol.check_routes_can_insert_and_get_sh_size(0);
  auto sh_size           = raft::alignTo(shared_route_size, sizeof(double)) +
                 sizeof(double) * sol.get_max_active_nodes_for_all_routes();

  if (!set_shmem_of_kernel(find_sliding_moves_tsp<i_t, f_t, REQUEST>, sh_size)) { return false; }

  find_sliding_moves_tsp<i_t, f_t, REQUEST>
    <<<n_blocks, n_threads, sh_size, sol.sol_handle->get_stream()>>>(
      sol.view(),
      move_candidates.view(),
      cuopt::make_span(sampled_tsp_data_),
      cuopt::make_span(locks_));
  RAFT_CHECK_CUDA(sol.sol_handle->get_stream());

  n_moves_found = thrust::count_if(sol.sol_handle->get_thrust_policy(),
                                   sampled_tsp_data_.begin(),
                                   sampled_tsp_data_.end(),
                                   is_sliding_tsp_initialized_t<i_t>());
  if (!n_moves_found) { return false; }

  sol.compute_max_active();
  moved_regions_.resize(sol.get_n_routes() * sol.get_max_active_nodes_for_all_routes(),
                        sol.sol_handle->get_stream());
  async_fill(moved_regions_, 1, sol.sol_handle->get_stream());

  set_moved_regions_kernel<i_t, f_t, REQUEST>
    <<<sol.get_n_routes(), 64, 0, sol.sol_handle->get_stream()>>>(sol.view(),
                                                                  cuopt::make_span(moved_regions_));
  RAFT_CHECK_CUDA(sol.sol_handle->get_stream());

  cuopt_func_call(
    move_candidates.debug_delta.set_value_to_zero_async(sol.sol_handle->get_stream()));
  cuopt_func_call(sol.compute_cost());
  cuopt_func_call(cost_before = is_tsp ? sol.get_total_cost(move_candidates.weights) -
                                           sol.get_cost(false, move_candidates.weights)
                                       : sol.get_cost(move_candidates.include_objective,
                                                      move_candidates.weights));

  auto size_of_frag = dimensions_route_t<i_t, f_t, REQUEST>::get_shared_size(
    sol.get_max_active_nodes_for_all_routes(), sol.problem_ptr->dimensions_info);
  sh_size = shared_route_size + size_of_frag;

  if (!set_shmem_of_kernel(execute_sliding_moves_tsp<i_t, f_t, REQUEST>, sh_size)) { return false; }

  thrust::sort(sol.sol_handle->get_thrust_policy(),
               sampled_tsp_data_.begin(),
               sampled_tsp_data_.end(),
               [] __device__(sliding_tsp_cand_t<i_t> cand1, sliding_tsp_cand_t<i_t> cand2) -> bool {
                 return cand1.selection_delta < cand2.selection_delta;
               });

  execute_sliding_moves_tsp<i_t, f_t, REQUEST>
    <<<sol.get_n_routes(), n_threads, sh_size, sol.sol_handle->get_stream()>>>(
      sol.view(),
      move_candidates.view(),
      cuopt::make_span(sampled_tsp_data_),
      cuopt::make_span(moved_regions_));
  RAFT_CHECK_CUDA(sol.sol_handle->get_stream());

  cuopt_func_call(sol.compute_cost());
  cuopt_func_call(
    cost_after = is_tsp ? sol.get_total_cost(move_candidates.weights) -
                            sol.get_cost(false, move_candidates.weights)
                        : sol.get_cost(move_candidates.include_objective, move_candidates.weights));

  cuopt_assert(abs((cost_before - cost_after) +
                     move_candidates.debug_delta.value(sol.sol_handle->get_stream()) <
                   EPSILON * (1 + abs(cost_before))),
               "Cost mismatch on sliding_tsp costs!");
  cuopt_assert(cost_before - cost_after >= EPSILON, "Cost should improve!");

  sol.global_runtime_checks(false, false, "sliding_tsp_end");
  return true;
}

template bool local_search_t<int, float, request_t::PDP>::perform_sliding_tsp(
  solution_t<int, float, request_t::PDP>& solution, move_candidates_t<int, float>& move_candidates);
template bool local_search_t<int, float, request_t::VRP>::perform_sliding_tsp(
  solution_t<int, float, request_t::VRP>& solution, move_candidates_t<int, float>& move_candidates);

}  // namespace detail
}  // namespace routing
}  // namespace cuopt
